#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/highgui/highgui.hpp"
#include <iostream>
#include <string>
#include <cstring>
#include <unistd.h>

using namespace std;
using namespace cv;

/// Constants
#define KERNEL_SIZE 5

/// Funcion Prototypes
void scalar_convolution(cv::Mat& source_image,
    cv::Mat& destiny_image,
    cv::Mat& kernel);

__global__ void scalar_convolution_oldschool(unsigned char* source_image,
    unsigned char* destiny_image,
    float* kernel,
    int k, 
    int w, int h, int channels);

int main( int argc, char** argv ) {

    // auxiliars
    Mat source_image, destiny_image, destiny_imageAux;
	unsigned char *source_image_raw, *destiny_image_raw;
    bool black_and_white = false, visual = false;
    bool dont_save_image = false, opencv_std = false;

    // validade the input
    if (argc < 2) {
        cerr <<
        "usage: ./smooth image_path [--black-and-white] [--opencv-std] \n\
                                    [--visual] [--dont-save-image] [--parallel]"
        << endl;
        return -1;
    }

    // check the optional values
	for (int i = 2; i < argc; i++) {
		if (string(argv[i]) == "--black-and-white") {
			black_and_white = true;
		}
		else if (string(argv[i]) == "--visual") {
			visual = true;
		}
		else if (string(argv[i]) == "--dont-save-image") {
			dont_save_image = true;
		}
	}

    // load the image and validate
    string image_path(argv[1]);

	if (!black_and_white) {
		source_image = imread(image_path, CV_LOAD_IMAGE_COLOR);
	} else {
		source_image = imread(image_path, CV_LOAD_IMAGE_GRAYSCALE);
	}

    if (!source_image.data) {
        cerr << "Abort! Couldn't load image!" << endl;
        return -1;
    }

    destiny_image = Mat::zeros(source_image.size (), source_image.type());
    destiny_imageAux = Mat::zeros(source_image.size (), source_image.type());

    if (opencv_std) {
        blur(source_image, destiny_image, Size(KERNEL_SIZE, KERNEL_SIZE));
    } else {

        // creates the kernel for a uniform filter
        Mat kernel = Mat::ones(KERNEL_SIZE, KERNEL_SIZE, CV_32F)/
            (float)(KERNEL_SIZE*KERNEL_SIZE);
		float *kernel_raw;


		// preparação para chamar o kernel CUDA
		size_t total_memory = source_image.total () * source_image.channels () * sizeof (unsigned char);
		size_t kernel_memory = kernel.total () * sizeof (float);
		cout << "Tamanho de memória total: " << total_memory << endl;
		cout << "Memória do Kernel: " << kernel_memory << " = " << kernel.total () << " * " << sizeof (float) << endl;

		if (hipMalloc (&source_image_raw, total_memory))	cerr << "Eita, hipMalloc falhou =/" << endl;
		if (hipMalloc (&destiny_image_raw, total_memory))	cerr << "Eita, hipMalloc falhou =/" << endl;
		if (hipMalloc (&kernel_raw, kernel_memory))		cerr << "Eita, hipMalloc falhou =/" << endl;

		hipMemcpy (source_image_raw, source_image.data, total_memory, hipMemcpyHostToDevice);
		hipMemcpy (destiny_image_raw, destiny_image.data, total_memory, hipMemcpyHostToDevice);
		hipMemcpy (kernel_raw, kernel.data, kernel_memory, hipMemcpyHostToDevice);

		// roda o normal
		scalar_convolution (source_image, destiny_imageAux, kernel);

        // e roda o kernel na GPU
        scalar_convolution_oldschool <<<1, 1>>> (
			source_image_raw, destiny_image_raw,
			kernel_raw, KERNEL_SIZE,
			source_image.cols, source_image.rows, source_image.channels ()
		);

		hipMemcpy (destiny_image.data, destiny_image_raw, total_memory, hipMemcpyDeviceToHost);

		unsigned char aux[total_memory];
		hipMemcpy (aux, destiny_image_raw, total_memory, hipMemcpyDeviceToHost);

		cout << "Resultado (entre Mats): " << memcmp (destiny_image.data, destiny_imageAux.data, total_memory) << endl;
		cout << "Resultado (entre raws): " << memcmp (aux, destiny_image.data, total_memory) << endl;

		hipFree (source_image_raw);
		hipFree (destiny_image_raw);
		hipFree (kernel_raw);
    }

	if (visual) {
		imshow("Original Image", source_image);
		imshow("Blured Image", destiny_image);
		waitKey(0);
	}

	if (!dont_save_image) {
		string new_file_name = "blured_" + image_path;
		imwrite(new_file_name, destiny_image);
	}

    return 0;
}

void scalar_convolution(cv::Mat& source_image,
    cv::Mat& destiny_image,
    cv::Mat& kernel) {

	int k = kernel.rows;
    int half_k = k / 2;
    int w = source_image.cols, h = source_image.rows;

	printf ("CPU - Imagem: %dx%d, kernel %dx%d de %f, número de canais %d\n", h, w, k, k, kernel.at<float> (0, 0), source_image.channels ());

    // performs convolution
    // for each pixel, either 1 channel (BW) or 3 channel (colored) images
	for (int channel = 0; channel < source_image.channels (); channel++) {
		for (int y = half_k; y < h - half_k; ++y) {
			for (int x = half_k; x < w - half_k; ++x) {

				float total = 0;

				// multiply the kernel values by all neighboors
				for (int i = -half_k; i <= half_k; ++i) {
					for (int j = -half_k; j <= half_k; ++j) {
						auto kernel_value = kernel.at<float> (i + half_k, j + half_k);

						auto pixel = source_image.ptr (y + i, x + j) + channel;
						total += (*pixel) * kernel_value;
					}
				}

				// the resulting pixel is the sum of the multiplications
				auto pixel = destiny_image.ptr (y, x) + channel;
				*pixel = total;
			}
		}
	}
}

__global__ void scalar_convolution_oldschool(unsigned char* source_image,
    unsigned char* destiny_image,
    float* kernel,
    int k, 
    int w, int h, int channels) {

	int half_k = k / 2;

	printf ("oi do %d %d\n", blockIdx.x, threadIdx.x);
	printf ("GPU - Imagem: %dx%d, kernel %dx%d de %f, número de canais %d\n", h, w, k, k, kernel[0], channels);

    // performs convolution
    // for each pixel, either 1 channel (BW) or 3 channel (colored) images
	for (int y = half_k; y < h - half_k; ++y) {
		for (int x = half_k; x < w - half_k; ++x) {
			for (int channel = 0; channel < channels; ++channel) {

				float total = 0;

				// multiply the kernel values by all neighboors
				for (int i = -half_k; i <= half_k; ++i) {
					for (int j = -half_k; j <= half_k; ++j) {
						auto kernel_value = kernel[i + half_k + (j + half_k) * k];

						auto pixel = source_image + channels * ((y + i) * w + x + j) + channel;
						total += *pixel * kernel_value;
					}
				}

				// the resulting pixel is the sum of the multiplications
				auto pixel = destiny_image + channels * (y * w + x) + channel;
				*pixel = total;
			}
		}
	}
}
